#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel_hash.cuh"
#include <array>
#include <random>
#include <iostream>
#include <algorithm>
#include <unordered_map>

__global__ void get_index(RH_hash_table<uint32_t, uint32_t> * d_hashtables, uint32_t* input_keys, uint32_t* input_vals){
    if(blockIdx.x < 1024)   {
        d_hashtables->lock();
        d_hashtables->insert(input_keys[blockIdx.x], input_vals[blockIdx.x]);
        d_hashtables->unlock();
    }
}
    
void unit_test(RH_hash_table<uint32_t, uint32_t>* h_hashtables) {
    // TODO
    // Generate test data
    std::random_device dev;
    std::mt19937 rng(dev());
    std::uniform_int_distribution<std::mt19937::result_type> dist(0, 127); // distribution in range [1, 6]
    
    std::array<uint32_t, 1024> numpool;
    std::array<uint32_t, 1024> valpool;
    std::unordered_map<uint32_t, uint32_t> cpu_hash;
    
    // Generate number pool
    for(int i = 0; i < numpool.size(); i++) {
        numpool[i] = i;
        valpool[i] = dist(rng);
    }
    
    std::shuffle(numpool.begin(), numpool.end(), rng);

    hipEvent_t start, stop;
    // Stopwatch
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Generate CPU answer
    for(int i = 0; i < numpool.size(); i++) {
        cpu_hash[numpool[i]] = valpool[i];
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "CPU Time elapsed: %3.1f ms\n", elapsedTime);

    // GPU
    uint32_t* d_numpool = NULL;
    uint32_t* d_valpool = NULL;
    gpuErrchk(hipMalloc((void**)&d_numpool, sizeof(uint32_t) * numpool.size()));
    gpuErrchk(hipMemcpy(d_numpool, numpool.data(), sizeof(uint32_t) * numpool.size(), hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc((void**)&d_valpool, sizeof(uint32_t) * numpool.size()));
    gpuErrchk(hipMemcpy(d_valpool, valpool.data(), sizeof(uint32_t) * valpool.size(), hipMemcpyHostToDevice));

    RH_hash_table<uint32_t, uint32_t> * d_hashtables = NULL;  
    gpuErrchk(hipMalloc((void**)&d_hashtables, sizeof(RH_hash_table<uint32_t, uint32_t>)));
    gpuErrchk(hipMemcpy(d_hashtables, h_hashtables, sizeof(RH_hash_table<uint32_t, uint32_t>), hipMemcpyHostToDevice));

    hipEventRecord(start, 0);
    // Does Actual work
    get_index<<<1024,1>>>(d_hashtables, d_numpool, d_valpool);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf( "GPU Time elapsed: %3.1f ms\n", elapsedTime);
    
    uint32_t *h_keys = new uint32_t[h_hashtables->capacity];
    uint32_t *h_vals = new uint32_t[h_hashtables->capacity];
    uint32_t *h_hash = new uint32_t[h_hashtables->capacity];

    gpuErrchk(hipMemcpy(h_keys, h_hashtables->buffer_keys, sizeof(uint32_t) * h_hashtables->capacity, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_vals, h_hashtables->buffer_values, sizeof(uint32_t) * h_hashtables->capacity, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_hash, h_hashtables->buffer_hash, sizeof(uint32_t) * h_hashtables->capacity, hipMemcpyDeviceToHost));

    for(int i = 0; i < h_hashtables->capacity; i++)  {
        if(h_hash[i] != 0)  {
            if(cpu_hash[h_keys[i]] != h_vals[i])    {
                printf("key = %d, val = %d\n", h_keys[i], h_vals[i]);
                fprintf(stderr,"Value does not match\n");    
            }
        }
    }
    delete [] h_keys;
    delete [] h_vals;
    delete [] h_hash;
}

int main() {
    RH_hash_table<uint32_t, uint32_t> h_hashtable;

    unit_test(&h_hashtable);

    return 0;
}
